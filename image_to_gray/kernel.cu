#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <string>
#include <cassert>
#include <stdio.h>
#include <ctime>

#include "stb_image.h"
#include "stb_image_write.h"

using namespace std;

struct Pixel
{
    unsigned char r, g, b, a;
};

__global__ void ConvertImageToGrayGpu(unsigned char* imageRGBA)
{
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t idx = y * blockDim.x * gridDim.x + x;

    Pixel* ptrPixel = (Pixel*)&imageRGBA[idx * 4];
    unsigned char pixelValue = (unsigned char)
        (ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
    ptrPixel->r = pixelValue;
    ptrPixel->g = pixelValue;
    ptrPixel->b = pixelValue;
    ptrPixel->a = 255;
}

int main(int argc, char** argv)
{
    // Check argument count
    if (argc < 2)
    {
        cout << "Usage: 02_ImageToGray <filename>";
        return -1;
    }

    // Open image
    int width, height, componentCount;
    cout << "Loading png file...";
    unsigned char* imageData = stbi_load(argv[1], &width, &height, &componentCount, 4);
    if (!imageData)
    {
        cout << endl << "Failed to open \"" << argv[1] << "\"";
        return -1;
    }
    cout << " DONE" << endl;

    // Validate image sizes
    if (width % 32 || height % 32)
    {
        // NOTE: Leaked memory of "imageData"
        cout << "Width and/or Height is not dividable by 32!";
        return -1;
    }

    //Start measuring time
    float elapsed1 = 0;
    hipEvent_t start1, stop1;

    hipEventCreate(&start1);
    hipEventCreate(&stop1);

    hipEventRecord(start1, 0);

    // Copy data to the gpu
    cout << "Copy data to GPU...";
    unsigned char* ptrImageDataGpu = nullptr;
    assert(hipMalloc(&ptrImageDataGpu, width * height * 4) == hipSuccess);
    assert(hipMemcpy(ptrImageDataGpu, imageData, width * height * 4, hipMemcpyHostToDevice) == hipSuccess);
    cout << " DONE" << endl;

    // Process image on gpu
    cout << "Running CUDA Kernel...";
    dim3 blockSize(32, 32);
    dim3 gridSize(width / blockSize.x, height / blockSize.y);
    ConvertImageToGrayGpu <<<gridSize, blockSize >>> (ptrImageDataGpu);
    auto err = hipGetLastError();
    cout << " DONE" << endl;

    // Copy data from the gpu
    cout << "Copy data from GPU...";
    assert(hipMemcpy(imageData, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost) == hipSuccess);
    cout << " DONE" << endl;

    // Stop measuring time and calculate the elapsed time
    hipEventRecord(stop1, 0);
    hipEventSynchronize(stop1);

    hipEventElapsedTime(&elapsed1, start1, stop1);

    hipEventDestroy(start1);
    hipEventDestroy(stop1);
    float time_1 = elapsed1;

    cout << "The elapsed time in gpu: " << time_1 << "ms" << endl;

    // Build output filename
    string fileNameOut = argv[1];
    fileNameOut = fileNameOut.substr(0, fileNameOut.find_last_of('.')) + "_gray1.png";

    // Write image back to disk
    cout << "Writing png to disk...";
    stbi_write_png(fileNameOut.c_str(), width, height, 4, imageData, 4 * width);
    cout << " DONE" << endl;

    // Free memory
    hipFree(ptrImageDataGpu);
    stbi_image_free(imageData);

    // Multiple GPUs

    // Check argument count
    if (argc < 2)
    {
        cout << "Usage: 02_ImageToGray <filename>";
        return -1;
    }

    // Open image
    int width2, height2, componentCount2;
    cout << "Loading png file...";
    unsigned char* imageData2 = stbi_load(argv[1], &width2, &height2, &componentCount2, 4);
    if (!imageData2)
    {
        cout << endl << "Failed to open \"" << argv[1] << "\"";
        return -1;
    }
    cout << " DONE" << endl;

    // Validate image sizes
    if (width2 % 32 || height2 % 32)
    {
        // NOTE: Leaked memory of "imageData"
        cout << "Width and/or Height is not dividable by 32!";
        return -1;
    }

    // Divide image into 4sub image
    const int divideImage2Width = width2/2;
    const int divideImage2Height = height2/2;

    // Alocate memory
    unsigned char* subImageData1 = new unsigned char[divideImage2Width * divideImage2Height * componentCount2];
    unsigned char* subImageData2 = new unsigned char[divideImage2Width * divideImage2Height * componentCount2];
    unsigned char* subImageData3 = new unsigned char[divideImage2Width * divideImage2Height * componentCount2];
    unsigned char* subImageData4 = new unsigned char[divideImage2Width * divideImage2Height * componentCount2];
    
    // Copy imageData2 to subImages
    for (int y = 0; y < divideImage2Height; y++) {
        for (int x = 0; x < divideImage2Width; x++) {
            for (int z = 0; z <= componentCount2; z++) {
                subImageData1[(y * divideImage2Width + x) * componentCount2 + z] = imageData2[(y * width2 + x) * componentCount2 + z];
                subImageData2[(y * divideImage2Width + x) * componentCount2 + z] = imageData2[(y * width2 + x + divideImage2Width) * componentCount2 + z];
                subImageData3[(y * divideImage2Width + x) * componentCount2 + z] = imageData2[((y + divideImage2Height) * width2 + x) * componentCount2 + z];
                subImageData4[(y * divideImage2Width + x) * componentCount2 + z] = imageData2[((y + divideImage2Height) * width2 + x + divideImage2Width) * componentCount2 + z];
            }
        }
    }

    // Create four CUDA streams.
    hipStream_t stream1; hipStreamCreate(&stream1);
    hipStream_t stream2; hipStreamCreate(&stream2);
    hipStream_t stream3; hipStreamCreate(&stream3);
    hipStream_t stream4; hipStreamCreate(&stream4);
    
    // Alocate GPU memory
    unsigned char* ptrSubImageDataGpu1 = nullptr;
    unsigned char* ptrSubImageDataGpu2 = nullptr;
    unsigned char* ptrSubImageDataGpu3 = nullptr;
    unsigned char* ptrSubImageDataGpu4 = nullptr;

    assert(hipMalloc(&ptrSubImageDataGpu1, divideImage2Width * divideImage2Height * 4) == hipSuccess);
    assert(hipMemcpy(ptrSubImageDataGpu1, subImageData1, divideImage2Width * divideImage2Height * 4, hipMemcpyHostToDevice) == hipSuccess);
    assert(hipMalloc(&ptrSubImageDataGpu2, divideImage2Width * divideImage2Height * 4) == hipSuccess);
    assert(hipMemcpy(ptrSubImageDataGpu2, subImageData2, divideImage2Width * divideImage2Height * 4, hipMemcpyHostToDevice) == hipSuccess);
    assert(hipMalloc(&ptrSubImageDataGpu3, divideImage2Width * divideImage2Height * 4) == hipSuccess);
    assert(hipMemcpy(ptrSubImageDataGpu3, subImageData3, divideImage2Width * divideImage2Height * 4, hipMemcpyHostToDevice) == hipSuccess);
    assert(hipMalloc(&ptrSubImageDataGpu4, divideImage2Width * divideImage2Height * 4) == hipSuccess);
    assert(hipMemcpy(ptrSubImageDataGpu4, subImageData4, divideImage2Width * divideImage2Height * 4, hipMemcpyHostToDevice) == hipSuccess);

    // Start measuring time
    float elapsed2 = 0;
    hipEvent_t start2, stop2;

    hipEventCreate(&start2);
    hipEventCreate(&stop2);

    hipEventRecord(start2, 0);

    // Copy data to the gpu
    cout << "Copy data to GPU...";
    unsigned char* ptrImageDataGpu2 = nullptr;
    assert(hipMemcpyAsync(ptrSubImageDataGpu1, subImageData1, divideImage2Width * divideImage2Height * 4, hipMemcpyHostToDevice, stream1) == hipSuccess);
    assert(hipMemcpyAsync(ptrSubImageDataGpu2, subImageData2, divideImage2Width * divideImage2Height * 4, hipMemcpyHostToDevice, stream2) == hipSuccess);
    assert(hipMemcpyAsync(ptrSubImageDataGpu3, subImageData3, divideImage2Width * divideImage2Height * 4, hipMemcpyHostToDevice, stream3) == hipSuccess);
    assert(hipMemcpyAsync(ptrSubImageDataGpu4, subImageData4, divideImage2Width * divideImage2Height * 4, hipMemcpyHostToDevice, stream4) == hipSuccess);
    cout << " DONE" << endl;

    // Process image on gpu
    cout << "Running CUDA Kernel...";
    dim3 blockSize2(32, 32);
    dim3 gridSize2(divideImage2Width / blockSize.x, divideImage2Height / blockSize.y);
    ConvertImageToGrayGpu <<<gridSize2, blockSize2, 0, stream1>>> (ptrSubImageDataGpu1);
    ConvertImageToGrayGpu <<<gridSize2, blockSize2, 0, stream2>>> (ptrSubImageDataGpu2);
    ConvertImageToGrayGpu <<<gridSize2, blockSize2, 0, stream3>>> (ptrSubImageDataGpu3);
    ConvertImageToGrayGpu <<<gridSize2, blockSize2, 0, stream4>>> (ptrSubImageDataGpu4);
    auto error = hipGetLastError();
    cout << " DONE" << endl;

    // Copy data from the gpu
    cout << "Copy data from GPU...";
    assert(hipMemcpy(subImageData1, ptrSubImageDataGpu1, divideImage2Width * divideImage2Height * 4, hipMemcpyDeviceToHost) == hipSuccess);
    assert(hipMemcpy(subImageData2, ptrSubImageDataGpu2, divideImage2Width * divideImage2Height * 4, hipMemcpyDeviceToHost) == hipSuccess);
    assert(hipMemcpy(subImageData3, ptrSubImageDataGpu3, divideImage2Width * divideImage2Height * 4, hipMemcpyDeviceToHost) == hipSuccess);
    assert(hipMemcpy(subImageData4, ptrSubImageDataGpu4, divideImage2Width * divideImage2Height * 4, hipMemcpyDeviceToHost) == hipSuccess);
    cout << " DONE" << endl;

    // Stop measuring time and calculate the elapsed time
    hipEventRecord(stop2, 0);
    hipEventSynchronize(stop2);

    hipEventElapsedTime(&elapsed2, start2, stop2);

    hipEventDestroy(start2);
    hipEventDestroy(stop2);
    float time_2 = elapsed2;

    cout << "The elapsed time in gpu: " << time_2 << "ms" << endl;

    // Build output filename
    for (int y = 0; y < divideImage2Height; y++) {
        for (int x = 0; x < divideImage2Width; x++) {
            for (int z = 0; z <= componentCount2; z++) {
                imageData2[(y * width2 + x) * componentCount2 + z] = subImageData1[(y * divideImage2Width + x) * componentCount2 + z];
                imageData2[(y * width2 + x + divideImage2Width) * componentCount2 + z] = subImageData2[(y * divideImage2Width + x) * componentCount2 + z];
                imageData2[((y + divideImage2Height) * width2 + x) * componentCount2 + z] = subImageData3[(y * divideImage2Width + x) * componentCount2 + z];
                imageData2[((y + divideImage2Height) * width2 + x + divideImage2Width) * componentCount2 + z] = subImageData4[(y * divideImage2Width + x) * componentCount2 + z];
            }
        }
    }

    string fileNameOut2 = argv[1];
    fileNameOut2 = fileNameOut2.substr(0, fileNameOut2.find_last_of('.')) + "_gray2.png";

    // Write image back to disk
    cout << "Writing png to disk...";
    stbi_write_png(fileNameOut2.c_str(), width2, height2, 4, imageData2, 4 * width2);
    cout << " DONE" << endl;

    // Free memory
    hipFree(ptrImageDataGpu2);
    stbi_image_free(imageData2);

    float time_d = time_2 - time_1;
    cout << "Time difference: " << time_d << "ms" << endl;
}