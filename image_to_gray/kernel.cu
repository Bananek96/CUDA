#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <string>
#include <cassert>
#include <chrono>
#include <stdio.h>
#include <time.h>

#include "stb_image.h"
#include "stb_image_write.h"

using namespace std;

struct Pixel
{
    unsigned char r, g, b, a;
};

__global__ void ConvertImageToGrayGpu(unsigned char* imageRGBA)
{
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t idx = y * blockDim.x * gridDim.x + x;

    Pixel* ptrPixel = (Pixel*)&imageRGBA[idx * 4];
    unsigned char pixelValue = (unsigned char)
        (ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
    ptrPixel->r = pixelValue;
    ptrPixel->g = pixelValue;
    ptrPixel->b = pixelValue;
    ptrPixel->a = 255;
}

float start_clock()
{
    float st_time = time(NULL);

    return st_time; 
}

float stop_clock(float st_time)
{
    float en_time = time(NULL);
    float time = (float)(en_time - st_time);

    return time;
}

int main(int argc, char** argv)
{
    // Start measuring time
    float st_time = start_clock();

    // Check argument count
    if (argc < 2)
    {
        cout << "Usage: 02_ImageToGray <filename>";
        return -1;
    }

    // Open image
    int width, height, componentCount;
    cout << "Loading png file...";
    unsigned char* imageData = stbi_load(argv[1], &width, &height, &componentCount, 4);
    if (!imageData)
    {
        cout << endl << "Failed to open \"" << argv[1] << "\"";
        return -1;
    }
    cout << " DONE" << endl;

    // Validate image sizes
    if (width % 32 || height % 32)
    {
        // NOTE: Leaked memory of "imageData"
        cout << "Width and/or Height is not dividable by 32!";
        return -1;
    }

    // Copy data to the gpu
    cout << "Copy data to GPU...";
    unsigned char* ptrImageDataGpu = nullptr;
    assert(hipMalloc(&ptrImageDataGpu, width * height * 4) == hipSuccess);
    assert(hipMemcpy(ptrImageDataGpu, imageData, width * height * 4, hipMemcpyHostToDevice) == hipSuccess);
    cout << " DONE" << endl;

    // Process image on gpu
    cout << "Running CUDA Kernel...";
    dim3 blockSize(32, 32);
    dim3 gridSize(width / blockSize.x, height / blockSize.y);
    ConvertImageToGrayGpu <<<gridSize, blockSize>>> (ptrImageDataGpu);
    auto err = hipGetLastError();
    cout << " DONE" << endl;

    // Copy data from the gpu
    cout << "Copy data from GPU...";
    assert(hipMemcpy(imageData, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost) == hipSuccess);
    cout << " DONE" << endl;

    // Build output filename
    string fileNameOut = argv[1];
    fileNameOut = fileNameOut.substr(0, fileNameOut.find_last_of('.')) + "_gray1.png";

    // Write image back to disk
    cout << "Writing png to disk...";
    stbi_write_png(fileNameOut.c_str(), width, height, 4, imageData, 4 * width);
    cout << " DONE" << endl;

    // Free memory
    hipFree(ptrImageDataGpu);
    stbi_image_free(imageData);
 
    // Stop measuring time and calculate the elapsed time
    float time_1 = stop_clock(st_time);

    printf("Time measured: %.3f seconds.\n", time_1);

    // Multiple GPUs

    // Start measuring time
    float st_time2 = start_clock();

    // Check argument count
    if (argc < 2)
    {
        cout << "Usage: 02_ImageToGray <filename>";
        return -1;
    }

    // Open image
    int width2, height2, componentCount2;
    cout << "Loading png file...";
    unsigned char* imageData2 = stbi_load(argv[1], &width2, &height2, &componentCount2, 4);
    if (!imageData2)
    {
        cout << endl << "Failed to open \"" << argv[1] << "\"";
        return -1;
    }
    cout << " DONE" << endl;

    // Validate image sizes
    if (width2 % 32 || height2 % 32)
    {
        // NOTE: Leaked memory of "imageData"
        cout << "Width and/or Height is not dividable by 32!";
        return -1;
    }

    // Create two CUDA streams.
    hipStream_t stream1; hipStreamCreate(&stream1);
    hipStream_t stream2; hipStreamCreate(&stream2);
    hipStream_t stream3; hipStreamCreate(&stream3);
    hipStream_t stream4; hipStreamCreate(&stream4);

    // Copy data to the gpu
    cout << "Copy data to GPU...";
    unsigned char* ptrImageDataGpu2 = nullptr;
    assert(hipMalloc(&ptrImageDataGpu2, width2 * height2 * 4) == hipSuccess);
    assert(hipMemcpyAsync(ptrImageDataGpu2, imageData2, width2 * height2 * 4, hipMemcpyHostToDevice, stream1) == hipSuccess);
    cout << " DONE" << endl;

    // Process image on gpu
    cout << "Running CUDA Kernel...";
    dim3 blockSize2(32, 32);
    dim3 gridSize2(width2 / blockSize.x, height2 / blockSize.y);
    ConvertImageToGrayGpu <<<gridSize2, blockSize2, 0 ,stream4 >>> (ptrImageDataGpu);
    ConvertImageToGrayGpu <<<gridSize2, blockSize2, 0, stream3 >>> (ptrImageDataGpu);
    ConvertImageToGrayGpu <<<gridSize2, blockSize2, 0, stream2 >>> (ptrImageDataGpu);
    ConvertImageToGrayGpu <<<gridSize2, blockSize2, 0, stream1 >>> (ptrImageDataGpu);
    auto error = hipGetLastError();
    cout << " DONE" << endl;

    // Copy data from the gpu
    cout << "Copy data from GPU...";
    assert(hipMemcpy(imageData2, ptrImageDataGpu2, width2 * height2 * 4, hipMemcpyDeviceToHost) == hipSuccess);
    cout << " DONE" << endl;

    // Build output filename
    string fileNameOut2 = argv[1];
    fileNameOut2 = fileNameOut2.substr(0, fileNameOut2.find_last_of('.')) + "_gray2.png";

    // Write image back to disk
    cout << "Writing png to disk...";
    stbi_write_png(fileNameOut2.c_str(), width2, height2, 4, imageData2, 4 * width2);
    cout << " DONE" << endl;

    // Free memory
    hipFree(ptrImageDataGpu2);
    stbi_image_free(imageData2);

    // Stop measuring time and calculate the elapsed time
    float time_2 = stop_clock(st_time2);

    printf("Time measured: %.3f seconds.\n", time_2);

    float  time_d = time_1 - time_2;
    printf("Time difference: %.3f seconds.\n", time_d);
}