#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <string>
#include <cassert>
#include <stdio.h>
#include <ctime>

#include "stb_image.h"
#include "stb_image_write.h"

using namespace std;

struct Pixel
{
    unsigned char r, g, b, a;
};

__global__ void ConvertImageToGrayGpu(unsigned char* imageRGBA)
{
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t idx = y * blockDim.x * gridDim.x + x;

    Pixel* ptrPixel = (Pixel*)&imageRGBA[idx * 4];
    unsigned char pixelValue = (unsigned char)
        (ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
    ptrPixel->r = pixelValue;
    ptrPixel->g = pixelValue;
    ptrPixel->b = pixelValue;
    ptrPixel->a = 255;
}

int main(int argc, char** argv)
{
    // Check argument count
    if (argc < 2)
    {
        cout << "Usage: 02_ImageToGray <filename>";
        return -1;
    }

    // Open image
    int width, height, componentCount;
    cout << "Loading png file...";
    unsigned char* imageData = stbi_load(argv[1], &width, &height, &componentCount, 4);
    if (!imageData)
    {
        cout << endl << "Failed to open \"" << argv[1] << "\"";
        return -1;
    }
    cout << " DONE" << endl;

    // Validate image sizes
    if (width % 32 || height % 32)
    {
        // NOTE: Leaked memory of "imageData"
        cout << "Width and/or Height is not dividable by 32!";
        return -1;
    }

    // Start measuring time
    float elapsed1 = 0;
    hipEvent_t start1, stop1;

    hipEventCreate(&start1);
    hipEventCreate(&stop1);

    hipEventRecord(start1, 0);

    // Copy data to the gpu
    cout << "Copy data to GPU...";
    unsigned char* ptrImageDataGpu = nullptr;
    assert(hipMalloc(&ptrImageDataGpu, width * height * 4) == hipSuccess);
    assert(hipMemcpy(ptrImageDataGpu, imageData, width * height * 4, hipMemcpyHostToDevice) == hipSuccess);
    cout << " DONE" << endl;

    // Process image on gpu
    cout << "Running CUDA Kernel...";
    dim3 blockSize(32, 32);
    dim3 gridSize(width / blockSize.x, height / blockSize.y);
    ConvertImageToGrayGpu <<<gridSize, blockSize>>> (ptrImageDataGpu);
    auto err = hipGetLastError();
    cout << " DONE" << endl;

    // Copy data from the gpu
    cout << "Copy data from GPU...";
    assert(hipMemcpy(imageData, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost) == hipSuccess);
    cout << " DONE" << endl;

    // Build output filename
    string fileNameOut = argv[1];
    fileNameOut = fileNameOut.substr(0, fileNameOut.find_last_of('.')) + "_gray1.png";

    // Stop measuring time and calculate the elapsed time
    hipEventRecord(stop1, 0);
    hipEventSynchronize(stop1);

    hipEventElapsedTime(&elapsed1, start1, stop1);

    hipEventDestroy(start1);
    hipEventDestroy(stop1);
    float time_1 = elapsed1;

    cout << "The elapsed time in gpu: " << time_1 << "ms" << endl;

    // Write image back to disk
    cout << "Writing png to disk...";
    stbi_write_png(fileNameOut.c_str(), width, height, 4, imageData, 4 * width);
    cout << " DONE" << endl;

    // Free memory
    hipFree(ptrImageDataGpu);
    stbi_image_free(imageData);

    // Multiple GPUs

    // Check argument count
    if (argc < 2)
    {
        cout << "Usage: 02_ImageToGray <filename>";
        return -1;
    }

    // Open image
    int width2, height2, componentCount2;
    cout << "Loading png file...";
    unsigned char* imageData2 = stbi_load(argv[1], &width2, &height2, &componentCount2, 4);
    if (!imageData2)
    {
        cout << endl << "Failed to open \"" << argv[1] << "\"";
        return -1;
    }
    cout << " DONE" << endl;

    // Validate image sizes
    if (width2 % 32 || height2 % 32)
    {
        // NOTE: Leaked memory of "imageData"
        cout << "Width and/or Height is not dividable by 32!";
        return -1;
    }

    // Making RGB layers

    // Create two CUDA streams.
    hipStream_t stream1; hipStreamCreate(&stream1);
    hipStream_t stream2; hipStreamCreate(&stream2);
    hipStream_t stream3; hipStreamCreate(&stream3);
    hipStream_t stream4; hipStreamCreate(&stream4);

    // Start measuring time
    float elapsed2 = 0;
    hipEvent_t start2, stop2;

    hipEventCreate(&start2);
    hipEventCreate(&stop2);

    hipEventRecord(start2, 0);

    // Copy data to the gpu
    cout << "Copy data to GPU...";
    unsigned char* ptrImageDataGpu2 = nullptr;
    assert(hipMalloc(&ptrImageDataGpu2, width2 * height2 * 4) == hipSuccess);
    assert(hipMemcpyAsync(ptrImageDataGpu2, imageData2, width2 * height2 * 4, hipMemcpyHostToDevice, stream1) == hipSuccess);
    assert(hipMemcpyAsync(ptrImageDataGpu2, imageData2, width2 * height2 * 4, hipMemcpyHostToDevice, stream2) == hipSuccess);
    assert(hipMemcpyAsync(ptrImageDataGpu2, imageData2, width2 * height2 * 4, hipMemcpyHostToDevice, stream3) == hipSuccess);
    assert(hipMemcpyAsync(ptrImageDataGpu2, imageData2, width2 * height2 * 4, hipMemcpyHostToDevice, stream4) == hipSuccess);
    cout << " DONE" << endl;

    // Process image on gpu
    cout << "Running CUDA Kernel...";
    dim3 blockSize2(32, 32);
    dim3 gridSize2(width2 / blockSize.x, height2 / blockSize.y);
    ConvertImageToGrayGpu <<<gridSize2, blockSize2, 0 ,stream4>>> (ptrImageDataGpu);
    ConvertImageToGrayGpu <<<gridSize2, blockSize2, 0, stream3>>> (ptrImageDataGpu);
    ConvertImageToGrayGpu <<<gridSize2, blockSize2, 0, stream2>>> (ptrImageDataGpu);
    ConvertImageToGrayGpu <<<gridSize2, blockSize2, 0, stream1>>> (ptrImageDataGpu);
    auto error = hipGetLastError();
    cout << " DONE" << endl;

    // Copy data from the gpu
    cout << "Copy data from GPU...";
    assert(hipMemcpy(imageData2, ptrImageDataGpu2, width2 * height2 * 4, hipMemcpyDeviceToHost) == hipSuccess);
    cout << " DONE" << endl;

    // Build output filename
    string fileNameOut2 = argv[1];
    fileNameOut2 = fileNameOut2.substr(0, fileNameOut2.find_last_of('.')) + "_gray2.png";

    // Stop measuring time and calculate the elapsed time
    hipEventRecord(stop2, 0);
    hipEventSynchronize(stop2);

    hipEventElapsedTime(&elapsed2, start2, stop2);

    hipEventDestroy(start2);
    hipEventDestroy(stop2);
    float time_2 = elapsed2;

    cout << "The elapsed time in gpu: " << time_2 << "ms" << endl;

    // Write image back to disk
    cout << "Writing png to disk...";
    stbi_write_png(fileNameOut2.c_str(), width2, height2, 4, imageData2, 4 * width2);
    cout << " DONE" << endl;

    // Free memory
    hipFree(ptrImageDataGpu2);
    stbi_image_free(imageData2);

    float time_d = time_2 - time_1;
    cout << "Time difference: " << time_d << "ms" << endl;
}